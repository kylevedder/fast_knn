#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <iostream>
#include <tuple>
#include <ratio>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include "mink.cuh"


// A chunk of work is blocksize-many points of P1.
// The number of potential chunks to do is N*(1+(P1-1)/blocksize)
// call (1+(P1-1)/blocksize) chunks_per_cloud
// These chunks are divided among the gridSize-many blocks.
// In block b, we work on chunks b, b+gridSize, b+2*gridSize etc .
// In chunk i, we work on cloud i/chunks_per_cloud on points starting from
// blocksize*(i%chunks_per_cloud).
template <typename scalar_t, int D, int K, typename MaxDist>
__global__ void KNearestNeighborKernelTruncated(
    const scalar_t* __restrict__ points1,
    const scalar_t* __restrict__ points2,
    const int64_t* __restrict__ lengths1,
    const int64_t* __restrict__ lengths2,
    scalar_t* __restrict__ dists,
    int64_t* __restrict__ idxs,
    const size_t P1,
    const size_t P2) {

  constexpr scalar_t max_dist = static_cast<scalar_t>(MaxDist::num / MaxDist::den);
  // Same idea as V2, but use register indexing for thread-local arrays.
  // Enabling sorting for this version leads to huge slowdowns; I suspect
  // that it forces min_dists into local memory rather than registers.
  // As a result this version is always unsorted.
  scalar_t cur_point[D];

  // Memory backing the RegisterMinK object.
  scalar_t min_dists[K];
  int min_idxs[K];
  const int64_t chunks_per_cloud = (1 + (P1 - 1) / blockDim.x);
  const int64_t chunks_to_do = chunks_per_cloud;
  for (int64_t chunk = blockIdx.x; chunk < chunks_to_do; chunk += gridDim.x) {
    const int64_t n = chunk / chunks_per_cloud;
    const int64_t start_point = blockDim.x * (chunk % chunks_per_cloud);
    int64_t p1 = start_point + threadIdx.x;
    if (p1 >= lengths1[n])
      continue;
    for (int d = 0; d < D; ++d) {
      cur_point[d] = points1[n * P1 * D + p1 * D + d];
    }
    int64_t length2 = lengths2[n];
    RegisterMinK<scalar_t, int, K> mink(min_dists, min_idxs);
    for (int p2 = 0; p2 < length2; ++p2) {
      scalar_t dist = 0;
      for (int d = 0; d < D; ++d) {
        int offset = n * P2 * D + p2 * D + d;
        scalar_t diff = cur_point[d] - points2[offset];
        scalar_t norm_diff = diff * diff;
        dist += norm_diff;
      }
      // if (dist >= max_dist) {
      //   continue;
      // }
      mink.add(dist, p2);
    }
    for (int k = 0; k < mink.size(); ++k) {
      idxs[n * P1 * K + p1 * K + k] = min_idxs[k];
      dists[n * P1 * K + p1 * K + k] = min_dists[k];
    }
  }
}

std::tuple<at::Tensor, at::Tensor> KNearestNeighborIdxCuda(
    const at::Tensor& p1,
    const at::Tensor& p2,
    const at::Tensor& lengths1,
    const at::Tensor& lengths2) {
  // Check inputs are on the same device
  at::TensorArg p1_t{p1, "p1", 1}, p2_t{p2, "p2", 2},
      lengths1_t{lengths1, "lengths1", 3}, lengths2_t{lengths2, "lengths2", 4};
  at::CheckedFrom c = "KNearestNeighborIdxCuda";
  at::checkAllSameGPU(c, {p1_t, p2_t, lengths1_t, lengths2_t});
  at::checkAllSameType(c, {p1_t, p2_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(p1.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();


  const auto P1 = p1.size(0);
  const auto P2 = p2.size(0);
  TORCH_CHECK(p1.size(1) == 3, "Point sets must have 3 dim");
  TORCH_CHECK(p2.size(1) == 3, "Point sets must have 3 dim");
  auto long_dtype = lengths1.options().dtype(at::kLong);
  // Backwards will skip points with idx -1
  auto idxs = at::full({P1}, -1, long_dtype);
  auto dists = at::zeros({P1}, p1.options());

  if (idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(idxs, dists);
  }

  const size_t threads = 256;
  const size_t blocks = 256;
  
  // MaxDist is the maximum distance that we will consider, in m^2.
  // Doing clown town ratio stuff because template must only take integers.
  using MaxDist = std::ratio<2, 1>;

  AT_DISPATCH_FLOATING_TYPES(p1.scalar_type(), "knn_kernel_cuda", ([&] {
                                KNearestNeighborKernelTruncated<scalar_t, 3, 1, MaxDist><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                                    p1.contiguous().data_ptr<scalar_t>(),
                                    p2.contiguous().data_ptr<scalar_t>(),
                                    lengths1.contiguous().data_ptr<int64_t>(),
                                    lengths2.contiguous().data_ptr<int64_t>(),
                                    dists.data_ptr<scalar_t>(),
                                    idxs.data_ptr<int64_t>(),
                                    P1,
                                    P2);
                              }));
  
  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(idxs, dists);
}

// ------------------------------------------------------------- //
//                   Backward Operators                          //
// ------------------------------------------------------------- //

// TODO(gkioxari) support all data types once AtomicAdd supports doubles.
// Currently, support is for floats only.
template <typename scalar_t>
__global__ void KNearestNeighborBackwardKernel(
    const scalar_t* __restrict__ p1, // (P1, 3)
    const scalar_t* __restrict__ p2, // (P2, 3)
    const int64_t* __restrict__ lengths1, // (1,)
    const int64_t* __restrict__ lengths2, // (1,)
    const int64_t* __restrict__ idxs, // (P1, 3)
    const scalar_t* __restrict__ grad_dists, // (P1, 3)
    scalar_t* __restrict__ grad_p1, // (P1, 3)
    scalar_t* __restrict__ grad_p2, // (P2, 3)
    const size_t P1,
    const size_t P2) {
  constexpr size_t D = 3;
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = gridDim.x * blockDim.x;

  for (size_t i = tid; i < P1 *  D; i += stride) {
    const size_t n = i / (P1 * D); // batch index
    size_t rem = i % (P1 * D);
    const size_t p1_idx = rem / D; // index of point in p1
    rem = rem % D;
    const size_t k = rem / D; // k-th nearest neighbor
    const size_t d = rem % D; // d-th dimension in the feature vector

    const size_t num1 = lengths1[n]; // number of valid points in p1 in batch
    const size_t num2 = lengths2[n]; // number of valid points in p2 in batch
    if ((p1_idx < num1) && (k < num2)) {
      const scalar_t grad_dist = grad_dists[n * P1 + p1_idx + k];
      // index of point in p2 corresponding to the k-th nearest neighbor
      const int64_t p2_idx = idxs[n * P1 + p1_idx + k];
      // If the index is the pad value of -1 then ignore it
      if (p2_idx == -1) {
        continue;
      }
      scalar_t diff = 2.0 * grad_dist *
            (p1[n * P1 * D + p1_idx * D + d] - p2[n * P2 * D + p2_idx * D + d]);
      atomicAdd(grad_p1 + n * P1 * D + p1_idx * D + d, diff);
      atomicAdd(grad_p2 + n * P2 * D + p2_idx * D + d, -1.0f * diff);
    }
  }
}

std::tuple<at::Tensor, at::Tensor> KNearestNeighborBackwardCuda(
    const at::Tensor& p1,
    const at::Tensor& p2,
    const at::Tensor& lengths1,
    const at::Tensor& lengths2,
    const at::Tensor& idxs,
    const at::Tensor& grad_dists) {
  // Check inputs are on the same device
  at::TensorArg p1_t{p1, "p1", 1}, p2_t{p2, "p2", 2},
      lengths1_t{lengths1, "lengths1", 3}, lengths2_t{lengths2, "lengths2", 4},
      idxs_t{idxs, "idxs", 5}, grad_dists_t{grad_dists, "grad_dists", 6};
  at::CheckedFrom c = "KNearestNeighborBackwardCuda";
  at::checkAllSameGPU(
      c, {p1_t, p2_t, lengths1_t, lengths2_t, idxs_t, grad_dists_t});
  at::checkAllSameType(c, {p1_t, p2_t, grad_dists_t});

  // This is nondeterministic because atomicAdd
  at::globalContext().alertNotDeterministic("KNearestNeighborBackwardCuda");

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(p1.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const auto P1 = p1.size(0);
  const auto P2 = p2.size(0);
  const auto D = 3;

  TORCH_CHECK(
      idxs.size(0) == P1, "KNN idxs must have the same point dimension as p1");
  TORCH_CHECK(grad_dists.size(0) == P1);

  auto grad_p1 = at::zeros({P1, D}, p1.options());
  auto grad_p2 = at::zeros({P2, D}, p2.options());

  if (grad_p1.numel() == 0 || grad_p2.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_p1, grad_p2);
  }

  const int blocks = 64;
  const int threads = 512;

  AT_DISPATCH_FLOATING_TYPES(p1.scalar_type(), "knn_kernel_cuda", ([&] {
    KNearestNeighborBackwardKernel<scalar_t><<<blocks, threads, 0, stream>>>(
        p1.contiguous().data_ptr<scalar_t>(),
        p2.contiguous().data_ptr<scalar_t>(),
        lengths1.contiguous().data_ptr<int64_t>(),
        lengths2.contiguous().data_ptr<int64_t>(),
        idxs.contiguous().data_ptr<int64_t>(),
        grad_dists.contiguous().data_ptr<scalar_t>(),
        grad_p1.data_ptr<scalar_t>(),
        grad_p2.data_ptr<scalar_t>(),
        P1,
        P2);
  }));

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(grad_p1, grad_p2);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("knn_forward", &KNearestNeighborIdxCuda, "KNN forward (CUDA)");
    m.def("knn_backward", &KNearestNeighborBackwardCuda, "KNN backward (CUDA)");
}
